#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include "network.h"
#include "detection_layer.h"
#include "cost_layer.h"
#include "utils.h"
#include "parser.h"
#include "box.h"
#include "image.h"
#include <sys/time.h>
}

/* Change class number here */
#define CLS_NUM 1

#ifdef OPENCV
#include "opencv2/highgui/highgui.hpp"
#include "opencv2/imgproc/imgproc.hpp"
extern "C" IplImage* image_to_Ipl(image img, int w, int h, int depth, int c, int step);
extern "C" image ipl_to_image(IplImage* src);
extern "C" void convert_yolo_detections(float *predictions, int classes, int num, int square, int side, int w, int h, float thresh, float **probs, box *boxes, int only_objectness);
extern "C" void draw_yolo(image im, int num, float thresh, box *boxes, float **probs);

extern "C" char *voc_names[];
extern "C" image voc_labels[];

static float **probs;
static box *boxes;
static network net;
static image in   ;
static image in_s ;
static image det  ;
static image det_s;
static image disp ;
static cv::VideoCapture cap;
static cv::VideoWriter cap_out;
static float fps = 0;
static float demo_thresh = 0;
static int w, h, depth, c, step= 0;
static int MODE = -1;
int NUMBER = 1;

void *fetch_in_thread(void *ptr)
{
    cv::Mat frame_m;
    cap >> frame_m;
    IplImage frame = frame_m;

if(step == 0)
{
    w = frame.width;
    h = frame.height;
    c = frame.nChannels;
    depth= frame.depth; 
    step = frame.widthStep;
}

    in = ipl_to_image(&frame);
    rgbgr_image(in);
    in_s = resize_image(in, net.w, net.h);
    return 0;
}

void *detect_in_thread(void *ptr)
{
    float nms = .4;

    detection_layer l = net.layers[net.n-1];
    float *X = det_s.data;
    float *predictions = network_predict(net, X);
    free_image(det_s);
    convert_yolo_detections(predictions, l.classes, l.n, l.sqrt, l.side, 1, 1, demo_thresh, probs, boxes, 0);
    if (nms > 0) do_nms(boxes, probs, l.side*l.side*l.n, l.classes, nms);
    printf("\033[2J");
    printf("\033[1;1H");
    printf("\nFPS:%.0f\n",fps);
    printf("Objects:\n\n");
    draw_detections(det, l.side*l.side*l.n, demo_thresh, boxes, probs, voc_names, voc_labels, CLS_NUM);

    if(MODE == 1)
    {
        IplImage* outputIpl= image_to_Ipl(det, w, h, depth, c, step);
        cv::Mat outputMat = cv::cvarrToMat(outputIpl, true);
        /*
        cvNamedWindow("image", CV_WINDOW_AUTOSIZE);
        cvShowImage("image", outputIpl); 
        cvWaitKey(1);  
        */
        cvReleaseImage(&outputIpl);
        cap_out << outputMat;
        outputMat.release();
     }

    return 0;
}

extern "C" void demo_yolo(char *cfgfile, char *weightfile, float thresh, int cam_index, char *videofile)
{
    demo_thresh = thresh;
    printf("YOLO demo\n");
    net = parse_network_cfg(cfgfile);
    if(weightfile){
        load_weights(&net, weightfile);
    }
    set_batch_network(&net, 1);

    srand(2222222);
printf("cam:%i\n",cam_index);
//videofile ="udpsrc multicast-iface=eth0 address=192.168.196.249 port=6000 ! application/x-rtp, media=(string)video, clockrate=(int)90000, encoding-name=(string)RAW, sampling=(string)YCbCr-4:2:2, depth=(string)8, width=(string)1280, height=(string)720 ! rtpvrawdepay ! videoconvert ! appsink";

//videofile = "udpsrc port=6000 ! application/x-rtp, encoding-name=H264,payload=96 ! rtph264depay ! ffdec_h264 ! appsink";
//"rtspsrc location=rtsp://192.168.99.98:8554/test ! rtph265depay ! h265parse ! openh265dec ! "

////videofile = "rtsp://192.168.99.98:8554/test";
if(cam_index != -1)
{
if (cam_index == -2){
MODE = 0; 
    cv::VideoCapture cam(videofile);
    cap = cam;
printf("opened!\n");

    if(!cap.isOpened()) error("Couldn't connect to webcam.\n");

}
else {

    MODE = 0; 
    cv::VideoCapture cam(cam_index);
    cap = cam;
    if(!cap.isOpened()) error("Couldn't connect to webcam.\n");
}

}
else 
{
    MODE = 1;
    printf("Video File name is: %s\n", videofile);
    cv::VideoCapture videoCap(videofile);
    cap = videoCap;
    if(!cap.isOpened()) error("Couldn't read video file.\n");

    cv::Size S = cv::Size((int)videoCap.get(CV_CAP_PROP_FRAME_WIDTH), (int)videoCap.get(CV_CAP_PROP_FRAME_HEIGHT));
    cv::VideoWriter outputVideo("out.avi", CV_FOURCC('D','I','V','X'), videoCap.get(CV_CAP_PROP_FPS), S, true);
    if(!outputVideo.isOpened()) error("Couldn't write video file.\n");
    cap_out = outputVideo;
}
 
    detection_layer l = net.layers[net.n-1];
    int j;

    boxes = (box *)calloc(l.side*l.side*l.n, sizeof(box));
    probs = (float **)calloc(l.side*l.side*l.n, sizeof(float *));
    for(j = 0; j < l.side*l.side*l.n; ++j) probs[j] = (float *)calloc(l.classes, sizeof(float *));

    pthread_t fetch_thread;
    pthread_t detect_thread;

    fetch_in_thread(0);
    det = in;
    det_s = in_s;

    fetch_in_thread(0);
    detect_in_thread(0);
				
				
				//char * test1 = new char[20], * test2= new char[20], *test3 = new char[20];
				
			
				

    disp = det;
    det = in;
    det_s = in_s;

    while(1){
        struct timeval tval_before, tval_after, tval_result;
        gettimeofday(&tval_before, NULL);
        if(pthread_create(&fetch_thread, 0, fetch_in_thread, 0)) error("Thread creation failed");
        if(pthread_create(&detect_thread, 0, detect_in_thread, 0)) error("Thread creation failed");

if (disp.w<=0&&disp.h<=0&&cam_index==-2){
//videofile = "udpsrc port=6000 ! application/x-rtp, encoding-name=H264,payload=96 ! rtph264depay ! ffdec_h264 ! appsink";
//videofile = "udpsrc port=6000 ! application/x-rtp, encoding-name=H265,payload=96 ! rtph265depay ! h265parse ! avdec_h265 !appsink ";
//videofile = "rtsp://192.168.196.10:554/Streaming/Channels/102";
 cv::VideoCapture cam(videofile);
    cap = cam;
printf("re-opened!\n");

    if(!cap.isOpened()) error("Couldn't connect to cam.\n");

}
        show_image(disp, "YOLO");
        free_image(disp);
        cvWaitKey(1);
        pthread_join(fetch_thread, 0);
        pthread_join(detect_thread, 0);

        disp  = det;
        det   = in;
        det_s = in_s;

     				//NUMBER++;
				//sprintf(test1, "kitti/%d.disp", NUMBER); sprintf(test2, "kitti/%d.det", NUMBER); sprintf(test3, "kitti/%d.det_s", NUMBER);
//sprintf(test, "%d.jpg", i);

       // cv::imwrite("kitti/1.png", det);
	//i++; sprintf(test, "kitti/%d.jpg", i);
//save_image_jpg(disp,test1);
//save_image_jpg(det,test2);
//save_image_jpg(det_s,test3);

	
        gettimeofday(&tval_after, NULL);
        timersub(&tval_after, &tval_before, &tval_result);
        float curr = 1000000.f/((long int)tval_result.tv_usec);
        fps = .9*fps + .1*curr;
    }
}
#else
extern "C" void demo_yolo(char *cfgfile, char *weightfile, float thresh, int cam_index){
    fprintf(stderr, "YOLO demo needs OpenCV for webcam images.\n");
}
#endif
